#include "hip/hip_runtime.h"
// MP Scan
// Given a list (lst) of length n
// Output its prefix sum = {lst[0], lst[0] + lst[1], lst[0] + lst[1] + ...
// +
// lst[n-1]}

#include <wb.h>

#define BLOCK_SIZE 512 //@@ You can change this

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

// Third kernel
// Input is output of first kernel
__global__ void scan_add(float *input, float *scan_block_sum, int len) {

    int i = 2 * (blockIdx.x * blockDim.x + threadIdx.x);
    if(blockIdx.x >= 1) {
        if(i < len)
            input[i] += scan_block_sum[blockIdx.x - 1];
        if(i + 1 < len)
            input[i + 1] += scan_block_sum[blockIdx.x - 1];
    }
}

__global__ void scan(float *input, float *output,float *auxiliary_arr, int len) {
  //@@ Modify the body of this function to complete the functionality of
  //@@ the scan on the device
  //@@ You may need multiple kernel calls; write your kernels before this
  //@@ function and call them from the host
    __shared__ float T[2 *  BLOCK_SIZE]; 
    // Loading shared mem
    int i = 2 * (blockIdx.x * blockDim.x + threadIdx.x);
    T[2 * threadIdx.x] = i < len ? input[i] : 0;
    T[2 * threadIdx.x + 1] = i + 1 < len ? input[i + 1] : 0;
    __syncthreads();

    // Reduction step
    int stride = 1;
    while(stride < 2 * BLOCK_SIZE) {
        __syncthreads();
        int index = (threadIdx.x + 1) * stride * 2 - 1;
        if(index < 2 * BLOCK_SIZE && (index - stride) >= 0)
            T[index] += T[index - stride];
        stride *= 2;
    }

    // Post scan step
    stride = BLOCK_SIZE / 2;
    while(stride > 0) {
        __syncthreads();
        int index = (threadIdx.x + 1) * stride * 2 - 1;
        if(index + stride < 2 * BLOCK_SIZE)
            T[index + stride] += T[index];
        stride /= 2;
    }

    __syncthreads();
    if (i < len)
        output[i] = T[2 * threadIdx.x];
    if (i + 1 < len)
        output[i + 1] = T[2 * threadIdx.x + 1];
    // Load auxiliary arry
    if (threadIdx.x + 1 == BLOCK_SIZE )
        auxiliary_arr[blockIdx.x] = T[2 * threadIdx.x + 1];
}

int main(int argc, char **argv) {
  wbArg_t args;
  float *hostInput;  // The input 1D list
  float *hostOutput; // The output list
  float *host_auxiliary;
  float *host_scan_block_sum;
  float *device_auxiliary;
  float *deviceInput;
  float *deviceOutput;
  float *device_scan_block_sum;
  float *device_place_holder; // Auxiliary arry for the second kernel call
  int numElements; // number of elements in the list

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostInput = (float *)wbImport(wbArg_getInputFile(args, 0), &numElements);
  int block_num = ceil(1.0 * numElements / (2.0 * BLOCK_SIZE));

  hostOutput = (float *)malloc(numElements * sizeof(float));
  host_auxiliary = (float *)malloc(block_num * sizeof(float));
  host_scan_block_sum = (float *)malloc(block_num * sizeof(float));
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The number of input elements in the input is ",
        numElements);

  wbTime_start(GPU, "Allocating GPU memory.");
  wbCheck(hipMalloc((void **)&deviceInput, numElements * sizeof(float)));
  wbCheck(hipMalloc((void **)&deviceOutput, numElements * sizeof(float)));
  wbCheck(hipMalloc((void **)&device_auxiliary, block_num * sizeof(float)));
  wbCheck(hipMalloc((void **)&device_scan_block_sum, block_num * sizeof(float)));
  wbCheck(hipMalloc((void **)&device_place_holder, sizeof(float)));
  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Clearing output memory.");
  wbCheck(hipMemset(deviceOutput, 0, numElements * sizeof(float)));
  wbTime_stop(GPU, "Clearing output memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  wbCheck(hipMemcpy(deviceInput, hostInput, numElements * sizeof(float),
                     hipMemcpyHostToDevice));
  wbTime_stop(GPU, "Copying input memory to the GPU.");

  //@@ Initialize the grid and block dimensions here
  dim3 DimGrid(block_num, 1, 1);
  dim3 DimBlock(BLOCK_SIZE, 1, 1);

  // -------------------------------------------------
  // ----------------------FIRST KERNEL---------------
  // -------------------------------------------------
  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Modify this to complete the functionality of the scan
  //@@ on the deivce
  scan<<<DimGrid, DimBlock>>>(deviceInput, deviceOutput,device_auxiliary,numElements);

  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  // wbCheck(hipMemcpy(host_auxiliary, device_auxiliary, block_num * sizeof(float),hipMemcpyDeviceToHost));
  // wbLog(TRACE, "block_num is ", block_num);
  // for(int i = 0; i < block_num; i++) {
  //       wbLog(TRACE, "auxiliary_arr item ", i, " is", host_auxiliary[i]);
  //   }

  // -------------------------------------------------
  // ----------------------SECOND KERNEL--------------
  // -------------------------------------------------
  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Modify this to complete the functionality of the scan
  //@@ on the deivce
  scan<<<1, ((block_num + 1) / 2)>>>(device_auxiliary, device_scan_block_sum,device_place_holder,block_num);

  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  // wbCheck(hipMemcpy(host_scan_block_sum, device_scan_block_sum, block_num * sizeof(float),hipMemcpyDeviceToHost));
  // wbLog(TRACE, "block_num is ", block_num);
  // for(int i = 0; i < block_num; i++) {
  //       wbLog(TRACE, "scan_block_sum item ", i, " is", host_scan_block_sum[i]);
  //   }

  // -------------------------------------------------
  // ----------------------THIRD KERNEL--------------
  // -------------------------------------------------
  scan_add<<<DimGrid, DimBlock>>>(deviceOutput,device_scan_block_sum, numElements);

  wbTime_start(Copy, "Copying output memory to the CPU");
  wbCheck(hipMemcpy(hostOutput, deviceOutput, numElements * sizeof(float),
                     hipMemcpyDeviceToHost));
  wbTime_stop(Copy, "Copying output memory to the CPU");



  wbTime_start(GPU, "Freeing GPU Memory");
  hipFree(deviceInput);
  hipFree(deviceOutput);
  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostOutput, numElements);

  // wbLog(TRACE, "Output[0] ",hostOutput[0]);
  // wbLog(TRACE, "Output[1] ",hostOutput[1]);
  // wbLog(TRACE, "Output[2] ",hostOutput[2]);
  // wbLog(TRACE, "Output[3] ",hostOutput[3]);
  free(hostInput);
  free(hostOutput);

  return 0;
}
