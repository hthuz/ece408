#include "hip/hip_runtime.h"
// MP Scan
// Given a list (lst) of length n
// Output its prefix sum = {lst[0], lst[0] + lst[1], lst[0] + lst[1] + ...
// +
// lst[n-1]}

#include <wb.h>

#define BLOCK_SIZE 1024 //@@ You can change this

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

__global__ void scan(float *input, float *output, int len) {
  //@@ Modify the body of this function to complete the functionality of
  //@@ the scan on the device
  //@@ You may need multiple kernel calls; write your kernels before this
  //@@ function and call them from the host
    __shared__ float T[2 *  BLOCK_SIZE]; 
    // Loading shared mem
    int i = 2 * (blockIdx.x * blockDim.x + threadIdx.x);
    T[2 * threadIdx.x] = i < len ? input[i] : 0;
    T[2 * threadIdx.x + 1] = i + 1 < len ? input[i + 1] : 0;
    __syncthreads();

    // Reduction step
    int stride = 1;
    while(stride < 2 * BLOCK_SIZE) {
        __syncthreads();
        int index = (threadIdx.x + 1) * stride * 2 - 1;
        if(index < 2 * BLOCK_SIZE && index - stride >= 0)
            T[index] += T[index - stride];
        stride *= 2;
    }

    // Post scan step
    stride = BLOCK_SIZE / 2;
    while(stride > 0) {
        __syncthreads();
        int index = (threadIdx.x + 1) * stride + 2 - 1;
        if(index + stride < 2 * BLOCK_SIZE)
            T[index + stride] += T[index];
        stride /= 2;
    }

    if (i < len)
        output[i] = T[2 * threadIdx.x];
    if (i + 1 < len)
        output[i + 1] = T[2 * threadIdx.x + 1];

}

int main(int argc, char **argv) {
  wbArg_t args;
  float *hostInput;  // The input 1D list
  float *hostOutput; // The output list
  float *deviceInput;
  float *deviceOutput;
  int numElements; // number of elements in the list

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostInput = (float *)wbImport(wbArg_getInputFile(args, 0), &numElements);
  hostOutput = (float *)malloc(numElements * sizeof(float));
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The number of input elements in the input is ",
        numElements);

  wbTime_start(GPU, "Allocating GPU memory.");
  wbCheck(hipMalloc((void **)&deviceInput, numElements * sizeof(float)));
  wbCheck(hipMalloc((void **)&deviceOutput, numElements * sizeof(float)));
  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Clearing output memory.");
  wbCheck(hipMemset(deviceOutput, 0, numElements * sizeof(float)));
  wbTime_stop(GPU, "Clearing output memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  wbCheck(hipMemcpy(deviceInput, hostInput, numElements * sizeof(float),
                     hipMemcpyHostToDevice));
  wbTime_stop(GPU, "Copying input memory to the GPU.");

  //@@ Initialize the grid and block dimensions here
  wbLog(TRACE, "DIM GRID: ", ceil(numElements * 1.0 / (2.0 * BLOCK_SIZE)));
  dim3 DimGrid(ceil(numElements * 1.0 / (2.0 * BLOCK_SIZE)), 1, 1);
  dim3 DimBlock(BLOCK_SIZE, 1, 1);

  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Modify this to complete the functionality of the scan
  //@@ on the deivce
  scan<<<DimGrid, DimBlock>>>(deviceInput, deviceOutput, numElements);

  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  wbCheck(hipMemcpy(hostOutput, deviceOutput, numElements * sizeof(float),
                     hipMemcpyDeviceToHost));
  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbTime_start(GPU, "Freeing GPU Memory");
  hipFree(deviceInput);
  hipFree(deviceOutput);
  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostOutput, numElements);

  wbLog(TRACE, "Output[0] ",hostOutput[0]);
  wbLog(TRACE, "Output[1] ",hostOutput[1]);
  wbLog(TRACE, "Output[2] ",hostOutput[2]);
  wbLog(TRACE, "Output[3] ",hostOutput[3]);
  free(hostInput);
  free(hostOutput);

  return 0;
}
