#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"

#define TILE_WIDTH 16

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
        std::cout<<"CUDA error: "<<hipGetErrorString(err)<<std::endl;  \
        exit(-1);                                                         \
    }                                                                     \
  } while (0)


/* convolution using tiled matrix multiplication */
/* X, Y dimension maps to matrix dimesion, Z is image number */
__global__ void conv_forward_kernel_multiply(float *output, const float *input, const float *mask, const int B, const int M, const int C, const int H, const int W, const int K,const int S)
{

    int H_out = (H - K) / S + 1;
    int W_out = (W - K) / S + 1;

    // A is mask, B is input_unrolled, C is multiplication result
    int numARows = M;
    int numAColumns = K * K * C;
    int numBRows = numAColumns;
    int numBColumns = H_out * W_out;
    int numCRows = numARows;
    int numCColumns = numBColumns;

    #define in_4d(i3, i2, i1, i0) input[(i3) * (C * H * W) + (i2) * (H * W) + (i1) * (W) + i0]

    int b = blockIdx.z;

    // Naive approach
    // int Row = blockIdx.y * blockDim.y + threadIdx.y;
    // int Col = blockIdx.x * blockDim.x + threadIdx.x;
    //
    // if(Row < numCRows && Col < numCColumns) 
    // {
    //     float Cval = 0;
    //     for(int i = 0; i < numAColumns; i++) {
    //         Cval += mask[Row * numAColumns + i] * 
    //             in_4d(b, i / (K * K), (Col / H_out) * S + (i % (K * K)) / K, (Col % H_out) * S + (i % (K * K)) % K);
    //         // Cval += mask[Row * numAColumns + i] * input[b * numBRows * numBColumns + i * numBColumns + Col];
    //     }
    //     output[b * numCRows * numCColumns + Row * numCColumns + Col] = Cval;
    // }

    // Tiled approach
    __shared__ float subTileA[TILE_WIDTH][TILE_WIDTH];
    __shared__ float subTileB[TILE_WIDTH][TILE_WIDTH];
    int bx = blockIdx.x; int by = blockIdx.y;
    int tx = threadIdx.x; int ty = threadIdx.y;
    int Row = by * TILE_WIDTH + ty;
    int Col = bx * TILE_WIDTH + tx;
    float Cval = 0;
    int i;

    for(int q = 0; q < (numAColumns - 1) / TILE_WIDTH + 1; q++) {
        if(Row < numARows && q * TILE_WIDTH + tx < numAColumns)
            subTileA[ty][tx] = mask[Row * numAColumns + q * TILE_WIDTH + tx];
        else
            subTileA[ty][tx] = 0;
        if(q * TILE_WIDTH + ty < numBRows && Col < numBColumns) {
            i = q * TILE_WIDTH + ty;
            subTileB[ty][tx] = in_4d(b, i / (K * K), (Col / H_out) * S + (i % (K * K)) / K, (Col % H_out) * S + (i % (K * K)) % K);
            // subTileB[ty][tx] = input[b * numBRows * numBColumns + (q * TILE_WIDTH + ty) * numBColumns + Col];
        }
        else
            subTileB[ty][tx] = 0;
        __syncthreads();
        if(Row < numCRows && Col < numCColumns) {
            for(int k = 0; k < TILE_WIDTH; k++)
                Cval += subTileA[ty][k] * subTileB[k][tx];
        }
        __syncthreads();
    }

    if(Row < numCRows && Col < numCColumns)
        output[b * numCRows * numCColumns + Row * numCColumns + Col] = Cval;

}
	
__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Allocate memory and copy over the relevant data structures to the GPU
    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;
    int num_input_elts = B * C * H * W;
    int num_output_elts = B * M * H_out * W_out;
    int num_mask_elts = M * C * K * K;

    wbCheck(hipMalloc((void**)device_input_ptr, num_input_elts * sizeof(float)));
    wbCheck(hipMalloc((void**)device_output_ptr, num_output_elts * sizeof(float)));
    wbCheck(hipMalloc((void**)device_mask_ptr, num_mask_elts * sizeof(float)));
    wbCheck(hipMemcpy(*device_input_ptr, host_input, num_input_elts * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(*device_mask_ptr, host_mask, num_mask_elts * sizeof(float), hipMemcpyHostToDevice));
   
}


__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Set the kernel dimensions and call the kernel
    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;
    int W_grid = ceil((W_out * 1.0) / (TILE_WIDTH * 1.0));
    int H_grid = ceil((H_out * 1.0) / (TILE_WIDTH * 1.0));

    // matrix multiplication
    dim3 gridDim_m(ceil((1.0 * H_out * W_out) / (1.0 * TILE_WIDTH)), ceil((1.0 * M) / (1.0 * TILE_WIDTH)),B);
    dim3 blockDim_m(TILE_WIDTH, TILE_WIDTH, 1);
    conv_forward_kernel_multiply<<<gridDim_m, blockDim_m>>>(device_output, device_input, device_mask, B, M, C, H, W, K, S);

}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Copy the output back to host
    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;
    int num_output_elts = B * M * H_out * W_out;
    wbCheck(hipMemcpy(host_output, device_output, num_output_elts * sizeof(float), hipMemcpyDeviceToHost));
    // Free device memory
    wbCheck(hipFree(device_input));
    wbCheck(hipFree(device_output));
    wbCheck(hipFree(device_mask));

}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}
