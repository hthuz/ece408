#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"

#define TILE_WIDTH 17

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
        std::cout<<"cuda error: "<<hipGetErrorString(err)<<std::endl;  \
        exit(-1);                                                         \
    }                                                                     \
  } while (0)

__constant__ float maskc[4096];

// __global__ void conv_forward_kernel(float *output, const float *input, const float *mask, const int B, const int M, const int C, const int H, const int W, const int K,const int S)
__global__ void conv_forward_kernel(float *output, const float *input, const int B, const int M, const int C, const int H, const int W, const int K,const int S)
{
    /*
    Function paramter definitions:
    output - output
    input - input
    mask - convolution kernel
    B - batch_size (number of images in x)
    M - number of output feature maps
    C - number of input feature maps
    H - input height dimension
    W - input width dimension
    K - kernel height and width (K x K)
    S - stride step length
    */

    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;
    // We have some nice #defs for you below to simplify indexing. Feel free to use them, or create your own.
    // An example use of these macros:
    // float a = in_4d(0,0,0,0)
    // out_4d(0,0,0,0) = a

    #define out_4d(i3, i2, i1, i0) output[(i3) * (M * H_out * W_out) + (i2) * (H_out * W_out) + (i1) * (W_out) + i0]
    #define in_4d(i3, i2, i1, i0) input[(i3) * (C * H * W) + (i2) * (H * W) + (i1) * (W) + i0]
    #define mask_4d(i3, i2, i1, i0) maskc[(i3) * (C * K * K) + (i2) * (K * K) + (i1) * (K) + i0]

    // Insert your GPU convolution kernel code here
    int W_grid = ceil((W_out * 1.0) / (TILE_WIDTH * 1.0));
    int H_grid = ceil((H_out * 1.0) / (TILE_WIDTH * 1.0));
    int m = blockIdx.x;
    int h = (blockIdx.y / W_grid) * TILE_WIDTH + threadIdx.y;
    int w = (blockIdx.y % W_grid) * TILE_WIDTH + threadIdx.x;
    int b = blockIdx.z;
    if(w < W_out && h < H_out) {
        float acc = 0.0f;
        for(int c = 0 ; c < C; c++) {
            for(int p = 0; p < K; p++) {
                for(int q = 0; q < K; q++) {
                    acc += in_4d(b, c, h * S + p, w * S + q) * mask_4d(m,c,p,q);
                }
            }
        }
        out_4d(b,m,h,w) = acc;
    }

    #undef out_4d
    #undef in_4d
    #undef mask_4d
}

	
__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Allocate memory and copy over the relevant data structures to the GPU
    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;
    int num_input_elts = B * C * H * W;
    int num_output_elts = B * M * H_out * W_out;
    int num_mask_elts = M * C * K * K;

    wbCheck(hipMalloc((void**)device_input_ptr, num_input_elts * sizeof(float)));
    wbCheck(hipMalloc((void**)device_output_ptr, num_output_elts * sizeof(float)));
    // wbCheck(hipMalloc((void**)device_mask_ptr, num_mask_elts * sizeof(float)));
    wbCheck(hipMemcpy(*device_input_ptr, host_input, num_input_elts * sizeof(float), hipMemcpyHostToDevice));
    // wbCheck(hipMemcpy(*device_mask_ptr, host_mask, num_mask_elts * sizeof(float), hipMemcpyHostToDevice));
    hipMemcpyToSymbol(HIP_SYMBOL(maskc), host_mask, num_mask_elts * sizeof(float), 0, hipMemcpyHostToDevice);
   
}


__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Set the kernel dimensions and call the kernel
    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;
    int W_grid = ceil((W_out * 1.0) / (TILE_WIDTH * 1.0));
    int H_grid = ceil((H_out * 1.0) / (TILE_WIDTH * 1.0));
    dim3 blockDim(TILE_WIDTH, TILE_WIDTH,1);
    dim3 gridDim(M, W_grid * H_grid, B);
    // conv_forward_kernel<<<gridDim, blockDim>>>(device_output, device_input, device_mask, B, M, C, H, W, K, S);
    conv_forward_kernel<<<gridDim, blockDim>>>(device_output, device_input, B, M, C, H, W, K, S);

}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Copy the output back to host
    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;
    int num_output_elts = B * M * H_out * W_out;
    wbCheck(hipMemcpy(host_output, device_output, num_output_elts * sizeof(float), hipMemcpyDeviceToHost));
    // Free device memory
    wbCheck(hipFree(device_input));
    wbCheck(hipFree(device_output));
    // wbCheck(hipFree(device_mask));

}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}
