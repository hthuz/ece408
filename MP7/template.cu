#include "hip/hip_runtime.h"
// Histogram Equalization

#include <wb.h>

#define HISTOGRAM_LENGTH 256
#define BLOCK_SIZE 256


//@@ insert code here
__global__ void float_2uchar(float *input , unsigned char* output, int size)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if(i < size)
        output[i] = (unsigned char)(255 * input[i]);
}

__global__ void rgb_2gray(unsigned char* input, unsigned char* output, int gsize, int imageChannels)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if(i < gsize)
        output[i] = (unsigned char)(0.21 * input[imageChannels * i] + 0.71 * input[imageChannels * i + 1] + 0.07 * input[imageChannels * i + 2]);
}

__global__ void histo_kernel(unsigned char* buffer, int gsize, unsigned int *histo)
{
    __shared__ unsigned int histo_private[HISTOGRAM_LENGTH];
    if(threadIdx.x < HISTOGRAM_LENGTH)
        histo_private[threadIdx.x] = 0;
    __syncthreads();

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    while(i < gsize) {
        atomicAdd(&(histo_private[buffer[i]]), 1);
        i += stride;
    }
    __syncthreads();

    if(threadIdx.x < HISTOGRAM_LENGTH)
        atomicAdd(&(histo[threadIdx.x]), histo_private[threadIdx.x]);

}

__global__ void scan(unsigned int *input, float *output, float *output_min, int len, int numPixels) {

    #define p(x) ((1.0 * x) / (1.0 * numPixels))

    __shared__ float T[2 *  BLOCK_SIZE]; 
    // Loading shared mem
    int i = 2 * (blockIdx.x * blockDim.x + threadIdx.x);
    T[2 * threadIdx.x] = i < len ? p(input[i]) : 0;
    T[2 * threadIdx.x + 1] = i + 1 < len ? p(input[i + 1]) : 0;
    __syncthreads();

    // Reduction step
    int stride = 1;
    while(stride < 2 * BLOCK_SIZE) {
        __syncthreads();
        int index = (threadIdx.x + 1) * stride * 2 - 1;
        if(index < 2 * BLOCK_SIZE && (index - stride) >= 0)
            T[index] += T[index - stride];
        stride *= 2;
    }

    // Post scan step
    stride = BLOCK_SIZE / 2;
    while(stride > 0) {
        __syncthreads();
        int index = (threadIdx.x + 1) * stride * 2 - 1;
        if(index + stride < 2 * BLOCK_SIZE)
            T[index + stride] += T[index];
        stride /= 2;
    }

    __syncthreads();
    if (i < len)
        output[i] = T[2 * threadIdx.x];
    if (i + 1 < len)
        output[i + 1] = T[2 * threadIdx.x + 1];

    if (threadIdx.x == 0)
        output_min[0] = p(input[0]);

    #undef p
}

__global__  void histo_equalization(unsigned char* input, unsigned char* output, float* CDF, float* CDFmin, int size)
{
    // #define clamp(x,start,end) (min(max(x, start), end))
    // #define correct_color(val) (clamp(255 * (CDF[val] - CDFmin[0]) / (1.0 - CDFmin[0]), 0.0, 255.0 ))
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < size)
        output[i] = (unsigned char) min(max(255 * (CDF[(int) input[i]] - CDF[0]) / (1.0 - CDF[0]),0.0),255.0);
        // output[i] = correct_color(input[i]);
    // #undef correct_color
    // #undef clamp
}

__global__ void uchar_2float(unsigned char *input , float* output, int size)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if(i < size)
        output[i] = (float) (input[i] / 255.0);
}

int main(int argc, char **argv) {
  wbArg_t args;
  int imageWidth;
  int imageHeight;
  int imageChannels;
  wbImage_t inputImage;
  wbImage_t outputImage;
  float *hostInputImageData;
  float *hostOutputImageData;
  const char *inputImageFile;

  //@@ Insert more code here

  args = wbArg_read(argc, argv); /* parse the input arguments */

  inputImageFile = wbArg_getInputFile(args, 0);

  wbTime_start(Generic, "Importing data and creating memory on host");
  inputImage = wbImport(inputImageFile);
  imageWidth = wbImage_getWidth(inputImage);
  imageHeight = wbImage_getHeight(inputImage);
  imageChannels = wbImage_getChannels(inputImage);
  outputImage = wbImage_new(imageWidth, imageHeight, imageChannels);
  hostInputImageData = wbImage_getData(inputImage);
  hostOutputImageData = wbImage_getData(outputImage);
  wbTime_stop(Generic, "Importing data and creating memory on host");

  //@@ insert code here

    // Step 1: float to uchar
    int size = imageWidth * imageHeight * imageChannels;
    int gsize = imageWidth * imageHeight; // Grayscale image size
    wbLog(TRACE, "size is ",size, " gray size is ", gsize);
    float* devInputImageData;
    unsigned char* devUcharImage;
    hipMalloc((void**)&devInputImageData, size * sizeof(float));
    hipMalloc((void**)&devUcharImage, size * sizeof(unsigned char));
    hipMemcpy(devInputImageData, hostInputImageData, size * sizeof(float), hipMemcpyHostToDevice);
    float_2uchar<<<ceil( (1.0 * size) / (1.0 * BLOCK_SIZE) ), BLOCK_SIZE>>>(devInputImageData, devUcharImage, size);

    // -------------------------------
    // PRINT
    // -------------------------------
    // unsigned char* hostUcharImage;
    // hostUcharImage = (unsigned char*) malloc(size * sizeof(unsigned char));
    // hipMemcpy(hostUcharImage, devUcharImage, size * sizeof(unsigned char), hipMemcpyDeviceToHost);
    // for(int i = size - 20; i < size; i++) {
    //     wbLog(TRACE, "floatin: ", hostInputImageData[i], " char: ", (int) hostUcharImage[i] );
    // }
    // -------------------------------
    // -------------------------------


    // Step 2: RGB to grayscale
    unsigned char* devGrayImage;
    hipMalloc((void**)&devGrayImage, gsize * sizeof(unsigned char));
    rgb_2gray<<<ceil( (1.0 * gsize) / (1.0 * BLOCK_SIZE) ), BLOCK_SIZE>>>(devUcharImage,devGrayImage,gsize, imageChannels);

    // -------------------------------
    // PRINT
    // -------------------------------
    // unsigned char* hostGrayImage;
    // hostGrayImage = (unsigned char*) malloc(size * sizeof(unsigned char));
    // hipMemcpy(hostGrayImage, devGrayImage, size * sizeof(unsigned char), hipMemcpyDeviceToHost);
    // for(int i = 0; i < 10; i++) {
    //     wbLog(TRACE, "gray image: ", (int) hostGrayImage[i]);
    // }
    // -------------------------------
    // -------------------------------
    
    // Step 3: Histogram of grayImage
    unsigned int* devHisto;
    hipMalloc((void**)&devHisto, HISTOGRAM_LENGTH * sizeof(int));
    hipMemset(devHisto, 0, HISTOGRAM_LENGTH * sizeof(int));
    histo_kernel<<<ceil( (1.0 * gsize) / (1.0 * BLOCK_SIZE) ), BLOCK_SIZE>>>(devGrayImage, gsize, devHisto);

    // -------------------------------
    // PRINT
    // -------------------------------
    // int* hostHisto;
    // hostHisto = (int*) malloc(size * sizeof(int));
    // hipMemcpy(hostHisto, devHisto, HISTOGRAM_LENGTH * sizeof(int), hipMemcpyDeviceToHost);
    // int sum = 0;
    // for(int i = 0; i < HISTOGRAM_LENGTH; i++) {
    //     wbLog(TRACE, i , " histo: ", hostHisto[i]);
    //     sum += hostHisto[i];
    // }
    // wbLog(TRACE, "Sum: ", sum);
    // -------------------------------
    // -------------------------------


    // Step 4: scan to compute cdf
    float* devCDF;
    float* devCDFmin;
    hipMalloc((void**)&devCDF, HISTOGRAM_LENGTH * sizeof(float));
    hipMalloc((void**)&devCDFmin, 1 * sizeof(float));
    scan<<<1,BLOCK_SIZE>>>(devHisto, devCDF,devCDFmin, HISTOGRAM_LENGTH, gsize );

    // -------------------------------
    // PRINT
    // -------------------------------
    // float* hostCDF;
    // float* hostCDFmin;
    // hostCDF = (float*) malloc(HISTOGRAM_LENGTH * sizeof(float));
    // hostCDFmin = (float*) malloc(1 * sizeof(float));
    // hipMemcpy(hostCDF, devCDF, HISTOGRAM_LENGTH * sizeof(float), hipMemcpyDeviceToHost);
    // hipMemcpy(hostCDFmin, devCDFmin, 1 * sizeof(float), hipMemcpyDeviceToHost);
    // for(int i = HISTOGRAM_LENGTH - 100; i < HISTOGRAM_LENGTH; i++) {
    //     wbLog(TRACE, i , " cdf: ", hostCDF[i]);
    // }
    // wbLog(TRACE, "CDFmin: ", hostCDFmin[0]);
    // -------------------------------
    // -------------------------------



    // Step 5: apply histogram equalization function
    unsigned char* devUcharImageCorrected;
    hipMalloc((void**)&devUcharImageCorrected, size * sizeof(unsigned char));
    histo_equalization<<<ceil( (1.0 * size) / (1.0 * BLOCK_SIZE) ),BLOCK_SIZE>>>(devUcharImage,devUcharImageCorrected,devCDF, devCDFmin, size);

    // -------------------------------
    // PRINT
    // -------------------------------
    // unsigned char* hostUcharImageCorrected;
    // hostUcharImageCorrected = (unsigned char*) malloc(size * sizeof(unsigned char));
    // hipMemcpy(hostUcharImageCorrected, devUcharImageCorrected, size * sizeof(unsigned char), hipMemcpyDeviceToHost);
    // for(int i = size - 20; i < size; i++) {
    //     wbLog(TRACE, i , " corrected: ", (int) hostUcharImageCorrected[i]);
    // }

    // -------------------------------
    // -------------------------------

    // Step 6: uchar to float
    float* devOutputImageData;
    hipMalloc((void**)&devOutputImageData, size * sizeof(float));
    uchar_2float<<<ceil( (1.0 * size) / (1.0 * BLOCK_SIZE) ), BLOCK_SIZE>>>(devUcharImageCorrected, devOutputImageData, size);
    hipMemcpy(hostOutputImageData, devOutputImageData, size * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    // -------------------------------
    // PRINT
    // -------------------------------
    // for(int i = 0; i < 200; i++) {
    //     wbLog(TRACE, i , " output: ", hostOutputImageData[i]);
    // }

    // -------------------------------
    // -------------------------------

    wbExport("outputimg.ppm", outputImage);

  wbSolution(args, outputImage);

  //@@ insert code here
    hipFree(devInputImageData);
    hipFree(devUcharImage);
    hipFree(devGrayImage);
    hipFree(devHisto);
    hipFree(devCDF);
    hipFree(devCDFmin);
    hipFree(devUcharImageCorrected);
    hipFree(devOutputImageData);

  return 0;
}
